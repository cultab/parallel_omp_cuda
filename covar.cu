#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>

#define PRINT 0
#define SIMPLE 0

// matrix size
#define SIZE_Y 2000
#define SIZE_X 2000

#define BLOCK_NUM 128
#define THREADS_NUM 128

// defines for the elem type
typedef double elem;

#ifdef __CUDA_ARCH__
#define syncthreads() __syncthreads()
#else
#define syncthreads()
#endif

inline void cudaPrintError(hipError_t cudaerr, const char *file, int line)
{
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "CUDA error: \"%s\" in file %s at line %d.\n", hipGetErrorString(cudaerr), file, line);
        exit(cudaerr);
    }
}

__host__ __device__ void print_mat(elem *mat, size_t height, size_t width, const char* name) {
    printf("%s (%ldx%ld):\n", name, height ,width);
    for (size_t i = 0; i < height; i++) {
        for (size_t j = 0; j < width; j++) {
            printf("%6.2f ", mat[i * width + j]);
        }
        printf("\n");
    }
}

#define cudaErr(ans)                                                                                                   \
    do {                                                                                                               \
        cudaPrintError((ans), __FILE__, __LINE__);                                                                     \
    } while (0)

#define cudaLastErr()                                                                                                  \
    do {                                                                                                               \
        hipError_t cudaerr = hipDeviceSynchronize();                                                                 \
        cudaPrintError(cudaerr, __FILE__, __LINE__);                                                                   \
    } while (0)

/*
 *
 * Calculate the average of each column,
 * subtract it from each element in that column.
 *
 * The calculation happens in-place, on the input matrix.
 *
 */
__global__ void col_average_distance_matrix(elem *d_A, size_t size_x, size_t size_y)
{
    elem col_average;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // printf("Thread(%d)\n", tid);

    int stride = gridDim.x * blockDim.x;


    for (int i = tid; i < size_x; i += stride) {
        col_average = d_A[0 * size_x + i];

        // find column average
        for (int j = 1; j < size_y; j++) {
            col_average += d_A[j * size_x + i];
        }

        col_average = col_average / (elem)size_y;

        // printf("Thread(%d): local [%d] = %f\n", thread_id, i, col_average);

        // subtract column average from each element of a column
        for (int j = 0; j < size_y; j++) {
            d_A[j * size_x + i] -= col_average;
        }
    }

}

/*
 * Matrix Multiplication of 2 matrices d_A and d_B.
 * The result is saved in d_Res.
 *
 * @row_A is the number of rows of d_A
 * @col_B is the number of columns of d_B
 * @col_A_common_row_B is the number of columns of d_A and the number of rows of d_B,
 *                     in other words it's d_A's and d_B's common dimension.
 *
 * Each thread computes multiple elements of the result.
 *
 * In our use case
 * the optimizations cause it to do approximatly (n^2/2)*n calculations (instead of n^3),
 * disregarding the diagonal,
 * where n is @col_A_common_row_B.
 *
 */
__global__ void matrix_mul(elem* d_A, elem* d_B, elem* d_Res, size_t row_A, size_t col_B, size_t col_A_common_row_B)
{
    // start element of each thread
    int start_x = threadIdx.x + blockIdx.x * blockDim.x; 
    int start_y = threadIdx.y + blockIdx.y * blockDim.y; 

    // stride of each thread
    int stride_x = gridDim.x;
    int stride_y = gridDim.y;

    // printf("tid(%d,%d): hre!\n", start_x, start_y);

    elem Pvalue;

    // each thread computes several elements of the output matrix
    for (int y = start_y; y < row_A; y += stride_y) {
        for (int x = start_x; x < col_B; x += stride_x) {
            // if it's a square matrix only compute the upper half triangle
            if (row_A == col_B && x < y) {
                continue;
            }
            Pvalue = 0;

            for (int k = 0; k < col_A_common_row_B; ++k) {
                // printf("Read from  A[%d][%d]\n"
                //        "Read from At[%d][%d]\n", y, k, k , y);
                Pvalue += d_A[y * col_A_common_row_B + k] * d_B[k * col_B + x];
            }

            // write back to the global memory
            d_Res[y* col_B + x] = Pvalue;
            // printf("Wrote %f to B[%d][%d]\n", Pvalue , y, x);

            // if it's a square matrix also save the Pvalue to the diagonally symmetric element
            if (row_A == col_B) {
                d_Res[x* col_B + y] = Pvalue;
            }
        }
    }
}
    // if (x == 0 && y == 0)
    // printf("%6.2f += %6.2f * %6.2f\n", Pvalue, d_A[y * com_col_A + k], d_B[k * col_B + x]);
    // printf("--------------------\n");


/* 
 * Transpose a matrix.
 *
 * @d_mat is the input matrix.
 * @d_result is the output matrix.
 *
 */

__global__ void matrix_transpose(elem* d_mat, elem* d_result, size_t size_y, size_t size_x)
{
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;

    int row_stride = gridDim.x;
    int col_stride = blockDim.x;

    // printf("Block(%d),Thread(%d): here!\n", block_id, thread_id);

    for (int i = block_id; i < size_y; i += row_stride) {
        for (int j = thread_id; j < size_x; j += col_stride) {
            d_result[j * size_y + i] = d_mat[i * size_x + j];
        }
    }
}

int main(void)
{
    size_t size_y = SIZE_Y;
    size_t size_x = SIZE_X;

    elem *A;
    elem *d_A;

    elem *At;
    elem *d_At;

    elem *B;
    elem *d_B;

    // Allocate host memory
    A = (elem*)malloc(size_y * size_x * sizeof(elem));
    if (A == NULL) {
        fprintf(stderr, "Failed to allocate memory at line %d\n", __LINE__);
        exit(-1);
    }
    At = (elem*)malloc(size_x * size_y * sizeof(elem));
    if (At == NULL) {
        fprintf(stderr, "Failed to allocate memory at line %d\n", __LINE__);
        exit(-1);
    }
    B = (elem*)malloc(size_y * size_y * sizeof(elem));
    if (B == NULL) {
        fprintf(stderr, "Failed to allocate memory at line %d\n", __LINE__);
        exit(-1);
    }

    hipEvent_t start, stop;
    float   elapsedTime;

    cudaErr(hipEventCreate(&start));
    cudaErr(hipEventCreate(&stop));


    cudaErr(hipMalloc(&d_A, size_y * size_x * sizeof(elem)));
    cudaErr(hipMalloc(&d_At, size_x * size_y * sizeof(elem)));
    cudaErr(hipMalloc(&d_B, size_y * size_y * sizeof(elem)));

    // initialize matrix A
    for (size_t i = 0; i < size_y; i++) {
        for (size_t j = 0; j < size_x; j++) {
            #if SIMPLE+0
                // A[i * size_x + j] = 10 * i + j;
                A[i * size_x + j] = i * j;
            #else
                A[i * size_x + j] = rand() % 10;
            #endif
            // A[i * size_x + j] = i * j;
        }
    }

    #if PRINT+0
    print_mat(A, size_y, size_x, "A");
    #endif

    // Copy A to device
    cudaErr(hipMemcpy(d_A, A, size_y * size_x * sizeof(elem), hipMemcpyHostToDevice));

    // start clock
    cudaErr(hipEventRecord(start, 0));

    // get new matrix
    col_average_distance_matrix<<<BLOCK_NUM, THREADS_NUM>>>(d_A, size_x, size_y);
    cudaLastErr();

    // stop clock
    cudaErr(hipEventRecord(stop, 0));
    cudaErr(hipEventSynchronize(stop));

    cudaErr(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time of first kernel:  %3.1f ms\n", elapsedTime);

    #if PRINT+0
    // copy A back to host (only to print)
    cudaErr(hipMemcpy(A, d_A, size_y * size_x * sizeof(elem), hipMemcpyDeviceToHost));
    #endif

    // transpose A into At
    matrix_transpose<<<BLOCK_NUM, THREADS_NUM>>>(d_A, d_At, size_y, size_x);
    cudaLastErr();

    //   32x32 =  1024  threads per block is the max
    // 256x256 = 65536 blocks per grid is the max (in order to support all compute capabilities)
    // dim3 dimBlock(32, 32);
    // dim3 dimGrid(256, 256);
    // dim3 dimBlock(1, 1);
    // dim3 dimGrid(1, 1);
    dim3 dimBlock(16, 16);
    dim3 dimGrid(256, 256);

    // start clock
    cudaErr(hipEventRecord(start, 0));

    // multiply A and At
    matrix_mul<<<dimGrid, dimBlock>>>(d_A, d_At, d_B, size_y, size_y, size_x);
    cudaLastErr();

    // stop clock
    cudaErr(hipEventRecord(stop, 0));
    cudaErr(hipEventSynchronize(stop));

    cudaErr(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time of second kernel:  %3.1f ms\n", elapsedTime);

    // copy result to host
    cudaErr(hipMemcpy(B, d_B, size_y * size_y * sizeof(elem), hipMemcpyDeviceToHost));

    #if PRINT+0
    // copy At back to host (only to print)
    cudaErr(hipMemcpy(At, d_At, size_y * size_x * sizeof(elem), hipMemcpyDeviceToHost));

    print_mat(A, size_y, size_x, "A - average of column");
    print_mat(At, size_x, size_y, "Transpose");
    print_mat(B, size_y, size_y, "Result");
    #endif

    // Free memory

    cudaErr(hipEventDestroy(start));
    cudaErr(hipEventDestroy(stop));

    cudaErr(hipFree(d_A));
    cudaErr(hipFree(d_At));
    cudaErr(hipFree(d_B));

    free(A);
    free(At);
    free(B);

    return 0;
}



// __global__ void MatrixMulKernel(elem* Md, elem* Nd, elem* Pd, size_t Width){{{
// {
//     printf("jere\n");
//     // declare cache in the shared memory
//     __shared__ elem Mds[BLOCK_DIM][BLOCK_DIM];
//     __shared__ elem Nds[BLOCK_DIM][BLOCK_DIM];
//
//     // keep track of column index of the Pd element using thread index
//     int x = threadIdx.x + blockIdx.x * blockDim.x; // x is column
//     // keep track of row index of the Pd element using thread index
//     int y = threadIdx.y + blockIdx.y * blockDim.y; // y is row
//
//     printf("Thread(%d,%d): here!\n", y, x);
//
//     // optimization ???
//     if (x < y) {
//         return;
//     }
//
//     elem Pvalue = 0;
//     // Loop over the Md and Nd block dimension required to compute the Pd element
//     for (int m = 0; m < Width/BLOCK_DIM; m++) {
//
//         // collaboratively loading of Md and Nd blocks into shared memory	 
//         Mds[threadIdx.y][threadIdx.x] = Md[y * Width + (m * BLOCK_DIM + threadIdx.x)];
//         Nds[threadIdx.y][threadIdx.x] = Md[(m * BLOCK_DIM + threadIdx.y) * Width + x];
//         syncthreads();
//
//         // keep track of the running sum
//         for (int k = 0; k < BLOCK_DIM; k++) {
//             Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
//         }
//         syncthreads();
//     }
//
//     // write back to the global memory
//     printf("Thread(%d,%d): Pvalue = %f\n", y, x, Pvalue);
//     Pd[y * Width + x] = Pvalue;
// }}}}




/*

    */
